#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "AES.h"
#include "main.h"

using namespace std;

int main(int argc, char **argv) {
	if(argc < 3) {
		printf("USAGE: aes KEY PLAINTEXT\n");
		return 1;
	}

	byte *key;
	uint ct[16], *pt;
	uint keySize = stringToByteArray(argv[1], &key);
	uint ptSize  = stringToByteArray(argv[2], &pt);

	if(keySize != 16 && keySize != 24 && keySize != 32) {
		printf("Invalid AES key size.\n");
		return 1;
	}

	if(ptSize != 4) {
		printf("Invalid AES block size.\n");
		return 1;
	}

	copyTables();

	AES *aes = new AES();
	aes->makeKey(key, keySize << 3, DIR_ENCRYPT);
	aes->encrypt(pt, ct);

	printHexArray(ct, 4);

	freeTables();

	return 0;
}

void copyTables() {
	int tableSize = 256*sizeof(uint);

	hipMalloc((void**)&cTe0, sizeof(Te0));
	hipMalloc((void**)&cTe1, sizeof(Te1));
	hipMalloc((void**)&cTe2, sizeof(Te2));
	hipMalloc((void**)&cTe3, sizeof(Te3));
	hipMalloc((void**)&cTe4, sizeof(Te4));

	hipMalloc((void**)&cTd0, sizeof(Td0));
	hipMalloc((void**)&cTd1, sizeof(Td1));
	hipMalloc((void**)&cTd2, sizeof(Td2));
	hipMalloc((void**)&cTd3, sizeof(Td3));
	hipMalloc((void**)&cTd4, sizeof(Td4));

	hipMemcpy(cTe0, Te0, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTe1, Te1, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTe2, Te2, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTe3, Te3, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTe4, Te4, tableSize, hipMemcpyHostToDevice);

	hipMemcpy(cTd0, Td0, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTd1, Td1, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTd2, Td2, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTd3, Td3, tableSize, hipMemcpyHostToDevice);
	hipMemcpy(cTd4, Td4, tableSize, hipMemcpyHostToDevice);
}

void freeTables() {
	hipFree(cTe0);
	hipFree(cTe1);
	hipFree(cTe2);
	hipFree(cTe3);
	hipFree(cTe4);

	hipFree(cTd0);
	hipFree(cTd1);
	hipFree(cTd2);
	hipFree(cTd3);
	hipFree(cTd4);
}

uint stringToByteArray(char *str, byte **array) {
	uint i, len  = strlen(str) >> 1;
	*array = (byte *)malloc(len * sizeof(byte));
	
	for(i=0; i<len; i++)
		sscanf(str + i*2, "%02X", *array+i);

	return len;
}

uint stringToByteArray(char *str, uint **array) {
	uint i, len  = strlen(str) >> 3;
	*array = (uint *)malloc(len * sizeof(uint));
	
	for(i=0; i<len; i++)
		sscanf(str + i*8, "%08X", *array+i);

	return len;
}

void printHexArray(uint *array, uint size) {
	uint i;
	for(i=0; i<size; i++)
		printf("%08X", array[i]);
	printf("\n");
}
