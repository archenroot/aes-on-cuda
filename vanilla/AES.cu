#include "hip/hip_runtime.h"
#include <stdio.h>
#include "AES.h"

#define GET(M,X,Y) ((M)[((Y) << 2) + (X)])

const uint size = 4*4*sizeof(uint);

void aes_encrypt_core(uint *cp, uint *cW, uint Nr) {
	uint i;
	
	AddRoundKey<<<1,16>>>(cp, cW);
	for(i=1; i<Nr; i++) {
		SubBytes<<<1,16>>>(cp);
		ShiftRows<<<1,4>>>(cp);
		MixColumns<<<1,4>>>(cp);
		AddRoundKey<<<1,16>>>(cp, cW+(i << 4));
	}
	SubBytes<<<1,16>>>(cp);
	ShiftRows<<<1,4>>>(cp);
	AddRoundKey<<<1,16>>>(cp, cW+(i << 4));
}

void aes_encrypt(uint *pt, uint *key, uint *ct, uint keysize) {
	uint *cp, *W, *cW, Nk, Nr;
	Nk = keysize >> 5;
	Nr = Nk + 6;

	uint s = ((Nr+1) * sizeof(uint)) << 4;
	W = (uint *)malloc(s);
	hipMalloc((void**)&cW, s);
	ExpandKeys(key, keysize, W, Nk, Nr);
	hipMemcpy(cW, W, s, hipMemcpyHostToDevice);

	hipMalloc((void**)&cp, size);
	hipMemcpy(cp, pt, size, hipMemcpyHostToDevice);

	aes_encrypt_core(cp, cW, Nr);

	hipMemcpy(ct, cp, size, hipMemcpyDeviceToHost);
}

void aes_encrypt_ecb(uint *pt, uint *key, uint *ct, uint keysize, uint n) {
	uint i, totalSize = n*size;

	uint *cp, *W, *cW, Nk, Nr;
	Nk = keysize >> 5;
	Nr = Nk + 6;

	uint s = ((Nr+1) * sizeof(uint)) << 4;
	W = (uint *)malloc(s);
	hipMalloc((void**)&cW, s);
	ExpandKeys(key, keysize, W, Nk, Nr);
	hipMemcpy(cW, W, s, hipMemcpyHostToDevice);

	hipMalloc((void**)&cp, totalSize);
	hipMemcpy(cp, pt, totalSize, hipMemcpyHostToDevice);
	
	for(i = 0; i < n; i++) {
		aes_encrypt_core(cp + (i << 4), cW, Nr);
	}

#ifndef NO_COPYBACK
	hipMemcpy(ct, cp, totalSize, hipMemcpyDeviceToHost);
#endif
}

void ExpandKeys(uint *key, uint keysize, uint *W, uint Nk, uint Nr) {
	uint i, j, cols, temp, tmp[4];
	cols = (Nr + 1) << 2;

	memcpy(W, key, (keysize >> 3)*sizeof(uint));

	for(i=Nk; i<cols; i++) {
		for(j=0; j<4; j++)
			tmp[j] = GET(W, j, i-1);
		if(Nk > 6) {
			if(i % Nk == 0) {
				temp   = hsbox[tmp[0]] ^  (Rcon[i/Nk] & 0x000000ff);
				tmp[0] = hsbox[tmp[1]] ^ ((Rcon[i/Nk] & 0xff000000) >> 24);
				tmp[1] = hsbox[tmp[2]] ^ ((Rcon[i/Nk] & 0x00ff0000) >> 16);
				tmp[2] = hsbox[tmp[3]] ^ ((Rcon[i/Nk] & 0x0000ff00) >>  8);
				tmp[3] = temp;
			} else if(i % Nk == 4) {
				tmp[0] = hsbox[tmp[0]];
				tmp[1] = hsbox[tmp[1]];
				tmp[2] = hsbox[tmp[2]];
				tmp[3] = hsbox[tmp[3]];
			}
		} else {
			if(i % Nk == 0) {
				temp   = hsbox[tmp[0]] ^  (Rcon[i/Nk] & 0x000000ff);
				tmp[0] = hsbox[tmp[1]] ^ ((Rcon[i/Nk] & 0xff000000) >> 24);
				tmp[1] = hsbox[tmp[2]] ^ ((Rcon[i/Nk] & 0x00ff0000) >> 16);
				tmp[2] = hsbox[tmp[3]] ^ ((Rcon[i/Nk] & 0x0000ff00) >>  8);
				tmp[3] = temp;
			}
		}
		for(j=0; j<4; j++)
			GET(W, j, i) = GET(W, j, i-Nk) ^ tmp[j];
	}
}

__global__ void SubBytes(uint *state) {
	uint i = threadIdx.x;
	state[i] = sbox[state[i]];
}

__global__ void ShiftRows(uint *state) {
	uint row  = threadIdx.x;
	uint i, tmp[4];

	for(i=0; i<4; i++)
		tmp[i] = state[row + 4*(i+row) % 16];
	for(i=0; i<4; i++)
		state[row + 4*i] = tmp[i];
}

#define xtime(x) ((x<<1) ^ (((x>>7) & 1) * 0x1b))
__global__ void MixColumns(uint *state) {
	uint col  = threadIdx.x;
	uint base = col << 2;
	uint t, Tmp, Tm;

	t   = state[base];
	Tmp = state[base] ^ state[base + 1] ^ state[base + 2] ^ state[base + 3];
	Tm  = state[base    ] ^ state[base + 1]; Tm = xtime(Tm) & 0xff; state[base    ] ^= Tm ^ Tmp;
	Tm  = state[base + 1] ^ state[base + 2]; Tm = xtime(Tm) & 0xff; state[base + 1] ^= Tm ^ Tmp;
	Tm  = state[base + 2] ^ state[base + 3]; Tm = xtime(Tm) & 0xff; state[base + 2] ^= Tm ^ Tmp;
	Tm  = state[base + 3] ^ t;               Tm = xtime(Tm) & 0xff; state[base + 3] ^= Tm ^ Tmp;
}

__device__ void AddRoundKey(uint *state, uint *key) {
	uint i = threadIdx.x;
	state[i] ^= key[i];
}
